#include <iostream>
#include <hip/hip_runtime.h>

using std::cout;
using std::endl;

__global__ void my_kernel()
{
    // I do absolutely nothing
}

int main(int argc, char *argv[])
{
    cout << "Hello world!! I will call a CUDA kernel now!!" << endl;
    my_kernel<<<1,1,0>>>();
    hipDeviceSynchronize();
    return 0;
}
