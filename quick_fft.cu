
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipfft/hipfft.h>

using std::cout;
using std::endl;

int main(int argc, char *argv[])
{

    hipfftComplex *sig1 = new hipfftComplex[8];
    hipfftComplex *sig2 = new hipfftComplex[8];
    hipfftComplex *sig3 = new hipfftComplex[8];

    for (int ii = 0; ii < 8; ii ++) {
        sig1[ii].x = 1.0f * (ii % 2);
        sig1[ii].y = 0.0f;
        sig2[ii].x = 0.0f;
        sig2[ii].y = -1.0f * (ii % 2);
        sig3[ii].x = sig1[ii].x;
        sig3[ii].y = sig2[ii].y;

    }

    hipfftComplex *d_s1;
    hipfftComplex *d_s2;
    hipfftComplex *d_s3;

    hipMalloc((void**)&d_s1, 8 * sizeof(hipfftComplex));
    hipMalloc((void**)&d_s2, 8 * sizeof(hipfftComplex));
    hipMalloc((void**)&d_s3, 8 * sizeof(hipfftComplex));

    hipMemcpy(d_s1, sig1, 8 * sizeof(hipfftComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_s2, sig2, 8 * sizeof(hipfftComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_s3, sig3, 8 * sizeof(hipfftComplex), hipMemcpyHostToDevice);

    hipfftHandle fftplan;
    hipfftPlan1d(&fftplan, 8, HIPFFT_C2C, 1);
    hipfftExecC2C(fftplan, d_s1, d_s1, HIPFFT_FORWARD);
    hipfftExecC2C(fftplan, d_s2, d_s2, HIPFFT_FORWARD);
    hipfftExecC2C(fftplan, d_s3, d_s3, HIPFFT_FORWARD);

    hipfftComplex *fft1 = new hipfftComplex[8];
    hipfftComplex *fft2 = new hipfftComplex[8];
    hipfftComplex *fft3 = new hipfftComplex[8];

    hipMemcpy(fft1, d_s1, 8 * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
    hipMemcpy(fft2, d_s2, 8 * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
    hipMemcpy(fft3, d_s3, 8 * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

    cout << "Signal 1: " << endl;
    for (int ii = 0; ii < 8; ii++) {
        cout << sig1[ii].x << " + i*" << sig1[ii].y << endl;
    }

    cout << "Signal 1 FFT: " << endl;
    for (int ii = 0; ii < 8; ii++) {
        cout << fft1[ii].x << " + i*" << fft1[ii].y << endl;
    }

    cout << "Signal 2: " << endl;
    for (int ii = 0; ii < 8; ii++) {
        cout << sig2[ii].x << " + i*" << sig2[ii].y << endl;
    }

    cout << "Signal 2 FFT: " << endl;
    for (int ii = 0; ii < 8; ii++) {
        cout << fft2[ii].x << " + i*" << fft2[ii].y << endl;
    }

    cout << "Signal 3: " << endl;
    for (int ii = 0; ii < 8; ii++) {
        cout << sig3[ii].x << " + i*" << sig3[ii].y << endl;
    }

    cout << "Signal 3 FFT: " << endl;
    for (int ii = 0; ii < 8; ii++) {
        cout << fft3[ii].x << " + i*" << fft3[ii].y << endl;
    }

    hipFree(d_s1);
    hipFree(d_s2);
    hipFree(d_s3);

    delete[] sig1;
    delete[] sig2;
    delete[] sig3;
    delete[] fft1;
    delete[] fft2;
    delete[] fft3;

    return 0;
}
