#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

using std::cout;
using std::endl;

__global__ void my_kernel()
{
    //cout << "Cout hello from the kernel!!" << endl;
    printf("Printf hello from the kernel!!\n");
    printf("I'm in thread %i\n", threadIdx.x);
}

int main(int argc, char *argv[])
{
    cout << "Hello world!! I will call a CUDA kernel now!!" << endl;
    my_kernel<<<1,1,0>>>();
    hipDeviceSynchronize();
    return 0;
}
