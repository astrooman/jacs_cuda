#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

using std::cout;
using std::endl;

__global__ add_me(int *a, int* b, int *c) {

    if(threadIdx.x < 8)
        c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];

}

int main(int argc, char *argv[])
{

    int arr1[8] = {1, 2, 3, 4 , 5 ,6 ,7, 8};
    int arr2[8] = {9, 10, 11, 12, 13, 14, 15 ,16};

    cout << "First array: " << endl;
    for (int x: arr1) {
        cout << x << endl;
    }

    cout << endl << "Second array: " << endl;
    for (int x: arr2) {
        cout << x << endl;
    }

    int *d_a;
    int *d_b;
    int *d_c;

    int h_sum[8];

    hipMalloc((void**)&d_a, 8 * sizeof(int));
    hipMalloc((void**)&d_b, 8 * sizeof(int));
    hipMalloc((void**)&d_c, 8 * sizeof(int));

    hipMemcpy(d_a, arr1, 8 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, arr2, 8 * sizeof(int), hipMemcpyHostToDevice);

    add_me<<<1, 16>>>(d_a, d_b, d_c);

    hipMemcpy(h_sum, d_c, 8 * sizeof(int), hipMemcpyDeviceToHost);

    cout << endl << "First array + second array: " << endl;

    for (int ii = 0; ii < 8; ii++)
        cout << h_sum[ii] << endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
