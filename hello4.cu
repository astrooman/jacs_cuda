#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

using std::cout;
using std::endl;

__global__ void my_kernel_1()
{

    printf("I'm in block %i, thread %i", blockIdx.x, threadIdx.x);

}

__global__ void my_kernel2()
{

    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = blockId * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    printf("Running thread %i in block %i", threadId, blockId);

    //prinft("Block position: x %i, y %i", blockIdx.x, blockIdx.y);
    //printf("Thread posirion: x %i, y %i", threadIdx.x, threadIdx.y);

}

int main(int argc, char *argv[])
{

    cout << "Hello world!! I will call the first CUDA kernel now!!" << endl;
    my_kernel_1<<<4, 4, 0>>>()

    dim3 nblocks(4, 1, 1);
    dim3 nthreads(4, 1, 1);
    cout << "Launching the second CUDA kernel now!!" << endl;
    my_kernel_1<<<nblocks, nthreads, 0>>>();

    dim3 nblocks2(2,2,1);
    dim3 nthreads2(2,2,1);
    cout << "Launching the third CUDA kernel now!!" << endl;
    my_kernel2<<<nblocks2, nthreads2, 0>>>();

    return 0;
}
