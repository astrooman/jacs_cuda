#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

using std::cout;
using std::endl;

__global__ void my_kernel_1()
{

    printf("I'm in block %i, thread %i\n", blockIdx.x, threadIdx.x);

}

__global__ void my_kernel2()
{

    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = blockId * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    printf("Running thread %i in block %i\n", threadId, blockId);

    printf("Block position: x %i, y %i\n", blockIdx.x, blockIdx.y);
    printf("Thread position: x %i, y %i\n", threadIdx.x, threadIdx.y);

}

int main(int argc, char *argv[])
{

    cout << "Hello world!! I will call the first CUDA kernel now!!" << endl;
    my_kernel_1<<<4, 4, 0>>>();

    dim3 nblocks(4, 1, 1);
    dim3 nthreads(4, 1, 1);
    cout << "Launching the second CUDA kernel now!!" << endl;
    my_kernel_1<<<nblocks, nthreads, 0>>>();
    hipDeviceSynchronize();

    dim3 nblocks2(2,2,1);
    dim3 nthreads2(2,2,1);
    cout << "Launching the third CUDA kernel now!!" << endl;
    my_kernel2<<<nblocks2, nthreads2, 0>>>();
    hipDeviceSynchronize();
    return 0;
}
