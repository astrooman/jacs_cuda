#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

using std::cout;
using std::endl;

__global__ void my_kernel(float mypi)
{
    printf("Printf hello from the kernel!!\n");
    printf("I'm in thread %i", threadIdx.x);
    printf("Someone sent me %d", mypi)
}

int main(int argc, char *argv[])
{
    cout << "Hello world!! I will call a CUDA kernel now!!" << endl;
    my_kernel<<<1,1,0>>>(3.1415f);

    return 0;
}
