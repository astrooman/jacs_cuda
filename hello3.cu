#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

using std::cout;
using std::endl;

__global__ void my_kernel(float mypi)
{
    printf("Printf hello from the kernel!!\n");
    printf("I'm in thread %i \n", threadIdx.x);
    printf("Someone sent me %f \n", mypi);
}

int main(int argc, char *argv[])
{
    cout << "Hello world!! I will call a CUDA kernel now!!" << endl;
    my_kernel<<<1,1,0>>>(3.1415f);
    hipDeviceSynchronize();
    return 0;
}
