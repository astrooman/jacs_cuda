#include <chrono>
#include <iostream>
#include <random>
#include <hip/hip_runtime.h>

using std::cout;
using std::endl;

__global__ void multiply_me_GPU(int *a, int *b, int *c, int width) {

    int row = blockIdx.y * gridDim.y + threadIdx.y;
    int column = blockIdx.x * gridDim.x + threadIdx.x;

    int sum = 0;

    for (int ii = 0; ii < width; ii++) {
        sum += a[row * width + ii] * b[column + ii * width];
    }

    c[row * width + column] = sum;

}

void multiply_me_CPU(int *a, int *b, int *c, int width) {

    int sum;

    for (int row = 0; row < width; row++) {
        for (int column = 0; column < width; column++) {
            sum = 0;
            for (int kk = 0; kk < width; kk++) {
                sum += a[row * width + kk] * b[column + kk * width];
            }
            c[row * width + column] = sum;
        }
    }

}


int main(int argc, char *argv[])
{

    int N = 1024;

    int *h_a = new int[N * N];
    int *h_b = new int[N * N];
    int *h_c = new int[N * N];
    int *h_c2 = new int [N * N];

    unsigned int seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::mt19937 engine(seed);
    std::uniform_int_distribution<int> distribution(0, 50);

    for (int ii = 0; ii < N * N; ii++) {
        h_a[ii] = distribution(engine);
        h_b[ii] = distribution(engine);
    }

    int *d_a;
    int *d_b;
    int *d_c;

    dim3 nblocks (32,32,1);
    dim3 nthreads(N/nblocks.x, N/nblocks.y, 1);

    float copy_elapsed;
    hipEvent_t copy_start;
    hipEvent_t copy_stop;
    hipEventCreate(&copy_start);
    hipEventCreate(&copy_stop);

    hipEventRecord(copy_start, 0);
    hipMalloc((void**)&d_a, N * N * sizeof(int));
    hipMalloc((void**)&d_b, N * N * sizeof(int));
    hipMalloc((void**)&d_c, N * N * sizeof(int));

    hipMemcpy(d_a, h_a, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(copy_stop, 0);
    hipEventSynchronize(copy_stop);
    hipEventElapsedTime(&copy_elapsed, copy_start, copy_stop);

    float GPU_elapsed;
    hipEvent_t GPU_start;
    hipEvent_t GPU_stop;
    hipEventCreate(&GPU_start);
    hipEventCreate(&GPU_stop);

    hipEventRecord(GPU_start, 0);
    multiply_me_GPU<<<nblocks, nthreads>>>(d_a, d_b, d_c, N);
    hipEventRecord(GPU_stop, 0);
    hipEventSynchronize(GPU_stop);
    hipEventElapsedTime(&GPU_elapsed, GPU_start, GPU_stop);

    hipMemcpy(h_c, d_c, N * N * sizeof(int), hipMemcpyDeviceToHost);

    std::chrono::time_point<std::chrono::system_clock> CPU_start, CPU_stop;
    std::chrono::duration<double> CPU_elapsed;

    CPU_start = std::chrono::system_clock::now();
    multiply_me_CPU(h_a, h_b, h_c2, N);
    CPU_stop = std::chrono::system_clock::now();
    CPU_elapsed = CPU_stop - CPU_start;

    cout << "It tool " << copy_elapsed / 1000.0f << "s to copy to data to the GPU" << endl;
    cout << "It took " << GPU_elapsed / 1000.0f << "s to multiply the matrix on the GPU" << endl;
    cout << "It took " << CPU_elapsed.count() << "s to multiply the matrix on the CPU" << endl;

    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    delete[] h_c2;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
