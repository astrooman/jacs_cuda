#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <hip/hip_runtime.h>

using std::cout;
using std::endl;

__global__ multiply_me_GPU(int *a, int *b, int *c) {


}

multiply_me_CPU(int *a, int *b, int *c) {

}


int main(int argc, char *argv[])
{

    int N = 2048;

    int *h_a = new int[N * N];
    int *h_b = new int[N * N];
    int *h_c = new int[N * N];
    int *h_c2 = new int [N * N];

    for (int ii = 0; ii < N; ii++) {
        h_a[ii] = ii;
        h_b[ii] = 2 * ii;
    }

    int *d_a;
    int *d_b;
    int *d_c;

    float copy_elapsed;
    hipEvent_t copy_start;
    hipEvent_t copy_stop;
    hipEventCreate(&copy_start);
    hipEventCreate(&copy_stop);

    hipEventRecord(copy_start, 0);
    hipMalloc((void**)&d_a, N * N * sizeof(int));
    hipMalloc((void**)&d_b, N * N * sizeof(int));
    hipMalloc((void**)&d_c, N * N * sizeof(int));

    hipMemcpy(d_a, h_a, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(copy_stop, 0);
    cudaEventSynchronise(copy_stop);
    cudaElapsedTime(&copy_elapsed, copy_start, copy_stop);

    float GPU_elapsed;
    hipEvent_t GPU_start;
    hipEvent_t GPU_stop;
    hipEventCreate(&GPU_start);
    hipEventCreate(&GPU_stop);

    hipEventRecord(GPU_start, 0);
    multiply_me_GPU<<<1, 256>>>(d_a, d_b, d_c);
    hipEventRecord(GPU_stop, 0);
    cudaEventSynchronise(GPU_stop);
    cudaElapsedTime(&GPU_elapsed, GPU_start, GPU_stop)

    hipMemcpy(h_c, d_c, N * N * sizeof(int), hipMemcpyDeviceToHost);

    std::chrono::time_point<std::chrono::system_clock> CPU_start, CPU_stop;
    std::chrono::duration<double> CPU_elapsed;

    CPU_start = std::chrono::system_clock::now();
    multiply_me_CPU(h_a, h_b, h_c2);
    CPU_stop = std::chrono::system_clock::now();
    CPU_elapsed = CPU_start - CPU_stop;

    cout << "It tool " << copy_elapsed / 1000.0f << "s to copy to data to the GPU" << endl;
    cout << "It took " << GPU_elapsed / 1000.0f << "s to multiply the matrix on the GPU" << endl;
    cout << "It took " << CPU_elapsed.count() << "s to multiply the matrix on the CPU" << endl;

    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    delete[] h_c2;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
